#include "hip/hip_runtime.h"
// raviteja meesala ## 06-02-01-10-51-14-1-11051
#include <stdio.h>
#include <assert.h>

#define N_t 1000
#define N_b 20
#define dimension 10000000
#define entries 30000000

 __global__ void mult_mat_vect(int *dev_row,int *dev_col,float *dev_ents,float *dev_vect,float *dev_res, int *dimer, int *N_tds, int *N_bls){

		int bid = blockIdx.x;
		int tid = threadIdx.x;

	int indx = bid*(*(N_tds)) + tid ; // Starts from zero and goes till as many threads deployed
	int dims = dimer[0]; // get the dimensions
	int i =0;
		if(indx < dims)
			dev_res[indx]= dev_vect[dev_col[dev_row[indx]]];

				while(indx < dims){
					dev_res[indx] = 0.0;
				for(i = dev_row[indx]; i< dev_row[indx+1];i++){
                                       dev_res[indx] = dev_res[indx] + dev_ents[i]*(dev_vect[dev_col[i]]);
				}

			indx = indx + (*(N_tds))*(*(N_bls)); // JUMP TO NEXT AVAILABLE ROW TO PICK;

			}	

}



				///////////////////////////////////////////////////////////////
					////////////	 Program main   ////////////
				//////////////////////////////////////////////////////////////


int main( int argc, char** argv)
{
	

					//////////////////////////////////////////
			////////////////////    HOST VARIABLES		 //////////////////////////////////
				/////////////////////////////////////////////////

	struct timespec start1, end1;
	double time_taken;

  int *row, *col;
  float *ents;
  float *vect;
  float *res;
	float* rest;
  int N_ts, N_bs;

	N_ts = N_t;
	N_bs = N_b; 

	int iter =0;

  int entry;
  int dim; 

  entry = entries;
  dim = dimension;


			//////////////////////////////////////////////////////////////////////////////////
				////////////////	DEVICE VARIABLES 	//////////////////
			//////////////////////////////////////////////////////////////////////////////////


   	int *dev_row, *dev_col;
	float *dev_ents;
	float *dev_vect;
	float *dev_res;
	int *N_tds,*N_bls; // No. of threads and blocks deployed
	int *dimer;
	int tp =0;
	int r_v = dim*(sizeof(int));
	int e_v = entry*(sizeof(int));

	int rd_v = dim*(sizeof(float));
	int ed_v = entry*(sizeof(float));
    // allocate device copies of a, b, c
    
					//////////////////////////////////////////////////////
			/////////////////////////// HOST VARIABLE ALLOCATIONS	//////////////////////////
					//////////////////////////////////////////////////

	row = (int *)malloc(r_v + sizeof(int));
	col = (int *)malloc(e_v);
	ents = (float *)malloc(ed_v);
	vect = (float *)malloc(rd_v);
	res = (float *)malloc(rd_v);
	rest= (float*)malloc(rd_v);

					///////////////////////////////////////////////////////
		///////////////////////////	HOST VARIABLE ASSIGNMENT 	////////////////////////////////
					//////////////////////////////////////////////////////


	row[0] = 0;

		for( tp = 1; tp<=dimension; tp++){
			row[tp] = row[tp-1] + 3;
	}

	col[0] = 0;
	col[1] = 1;
	col[2] = 2;

	ents[0] = 1;
	ents[1] = 2;
	ents[2] = 1;
	iter = 3;
	
		for( tp = 1; tp< dimension-1; tp++){

			col[tp*3] =tp -1;
			col[tp*3 + 1] = tp;  
			col[tp*3 + 2] = tp+1;
			ents[tp*3] = 1;
			ents[tp*3 + 1] = 2;
			ents[tp*3 + 2] = 1;
		//	iter = tp
		}

	
		col[tp*3] = tp-2;
		col[tp*3 + 1] = tp-1;
		col[tp*3 +2] = tp;
		ents[tp*3] = 1;
		ents[tp*3 +1] = 2;
		ents[tp*3 +2] = 1;

//				printf("%d heybro! %d", tp*3 +3, tp*3 +3);
	
		for( tp = 0; tp<dimension; tp++){
			vect[tp] = tp/1000;
			}


/////////////////////////////////////////////////////					///////////////////////


//	for(tp = 0; tp<=dimension; tp=tp+(entries/100))	
//		printf("row: %d, vect: %f \n", row[tp],vect[tp]);

//	for(tp=0; tp<entries;tp= tp+(entries/100))
//		printf("col:%d val:%f \n", col[tp],ents[tp]);


//////////////////////////////////////////	//////////////////////////////////////	/////////////////////////////////////
					/////////////////////////////////////////////////////	
			///////////////////////////	DEVICE ALLOCATIONS 	//////////////////////////
					//////////////////////////////////////////////////


	hipMalloc((void**)&dev_row,r_v + sizeof(int) );
	hipMalloc((void**)&dev_col, e_v);
	hipMalloc((void**)&dev_ents,ed_v);
	hipMalloc((void**)&dev_vect, rd_v);
	hipMalloc((void**)&dev_res, rd_v);

	hipMalloc((void**)&N_tds, sizeof(int));	
	hipMalloc((void**)&N_bls, sizeof(int));
	hipMalloc((void**)&dimer, sizeof(int));

hipDeviceSynchronize();
					/////////////////////////////////////////////////////
			/////////////////////	 COPY FROM HOST TO DEVICE   ///////////////////////////
					////////////////////////////////////////////////////////

	hipMemcpy(dev_row, row, r_v + sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_col, col, e_v, hipMemcpyHostToDevice);
	hipMemcpy(dev_ents, ents, ed_v, hipMemcpyHostToDevice);
	hipMemcpy(dev_vect, vect, rd_v, hipMemcpyHostToDevice);
	
	hipMemcpy(N_tds,&N_ts, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(N_bls,&N_bs, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dimer,&dim, sizeof(int), hipMemcpyHostToDevice);

hipDeviceSynchronize();

clock_gettime(CLOCK_MONOTONIC,&start1);
				//////////////////////////////////////////////////////////////
			//////////////////#########	CALL KERNEL 	##########///////////////////////////
				/////////////////////////////////////////////////////////////


	mult_mat_vect<<<N_b,N_t>>>( dev_row, dev_col, dev_ents, dev_vect, dev_res,dimer, N_tds, N_bls);

	hipDeviceSynchronize();

clock_gettime(CLOCK_MONOTONIC, &end1);

time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);

	printf("Non sequential time: %lf \n", time_taken);

				/////////////////////////////////////////////////////////////
			/////////////////////		COPY RESULTS 	////////////////////////////////////////////
				//////////////////////////////////////////////////////////

	hipMemcpy(res, dev_res, rd_v, hipMemcpyDeviceToHost);

hipDeviceSynchronize();

//	for(tp =0; tp<dimension;tp+= (dimension/100))
//	printf("value: %d, %f \n",tp,res[tp]);


	hipFree(dev_row);
	hipFree(dev_col);
	hipFree(dev_ents);
	hipFree(dev_vect);
	hipFree(dev_res);

clock_gettime(CLOCK_MONOTONIC,&start1);

				////////////////////////////////////////////////////////////////////////////	
		//////////////////////////////		SEQUENTIAL 	CODE		////////////////////////////////////	
				///////////////////////////////////////////////////////////////////////////

	for(tp = 0; tp<dimension;tp++){
			rest[tp] = 0;
		for(iter = row[tp]; iter<row[tp+1]; iter++){
			rest[tp] = rest[tp] + (ents[iter]*(vect[col[iter]]));					
			

		}	
	}


clock_gettime(CLOCK_MONOTONIC, &end1);

time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);

        printf("Sequential time: %lf \n", time_taken);
//        for(tp =0; tp<dimension;tp+= (dimension/100))
  //      printf("value difference: %d, %f \n",tp,rest[tp] - res[tp]);



    return 0;
}
