#include "hip/hip_runtime.h"
// Meesala Raviteja
// 06-02-01-10-51-14-1-11051
#include <stdio.h>
#include <assert.h>

#define N_t 64
#define N_b 100
#define rows 6400
#define columns 6400

 __global__ void mult_mat_vectdiag(float *dev_A,float *dev_b, float *dev_c, int *row_k, int *col_k, int *N_bls,int *N_tds){

		int bid = blockIdx.x;
		int tid = threadIdx.x;
        int i=0;
        int c_k = bid*(*N_tds) + tid;
        
        for(i=0;i< *(row_k);i++)
            *(dev_c + c_k) = *(dev_c + c_k) +  *(dev_A + i*(*row_k) + c_k)*(*(dev_b+(c_k+i)%rows));
}

 __global__ void mult_mat_vectrow(float *dev_A,float *dev_b, float *dev_c, int *row_k, int *col_k, int *N_bls,int *N_tds){

		int bid = blockIdx.x;
		int tid = threadIdx.x;
        int i=0;
        int c_k = bid*(*N_tds) + tid;
        
        for(i=0;i< *(row_k);i++)
            *(dev_c + i) = *(dev_c + i) +  *(dev_A + i*(*row_k) + c_k)*(*(dev_b+c_k));
}

 __global__ void mult_mat_vectcol(float *dev_A,float *dev_b, float *dev_c, int *row_k, int *col_k, int *N_bls,int *N_tds){

		int bid = blockIdx.x;
		int tid = threadIdx.x;
        int i=0;
        int c_k = bid*(*N_tds) + tid;
        
        for(i=0;i< *(row_k);i++)
            *(dev_c + i) = *(dev_c + i) +  *(dev_A + i+ c_k*(*row_k) )*(*(dev_b+c_k));
}
				///////////////////////////////////////////////////////////////
					////////////	 Program main   ////////////
				//////////////////////////////////////////////////////////////

int main( int argc, char** argv)
{	
					//////////////////////////////////////////
			////////////////////    HOST VARIABLES		 //////////////////////////////////
				/////////////////////////////////////////////////

	struct timespec start1, end1;
	double time_taken;

    float *A, *b, *c;
    int i,j; 
    int N_ts, N_bs;
    int r_s=rows;
    int c_s = columns;
    N_ts = N_t;
	N_bs = N_b;

			//////////////////////////////////////////////////////////////////////////////////
				////////////////	DEVICE VARIABLES 	//////////////////
			//////////////////////////////////////////////////////////////////////////////////

   	float *dev_A, *dev_b,*dev_c;
    int *dev_rs,*dev_cs;
	int *N_tds,*N_bls; // No. of threads and blocks deployed
	    
					//////////////////////////////////////////////////////
			/////////////////////////// HOST VARIABLE ALLOCATIONS	//////////////////////////
					//////////////////////////////////////////////////

	A = (float *)malloc(sizeof(float)*rows*columns);
	b = (float *)malloc(sizeof(float)*columns);
	c = (float *)malloc(sizeof(float)*rows);
	
					///////////////////////////////////////////////////////
		///////////////////////////	HOST VARIABLE ASSIGNMENT 	////////////////////////////////
					//////////////////////////////////////////////////////

        for(i=0;i<rows;i++){
            c[i] = 0.0;   
            for(j=0;j<columns;j++)
	            *(A + i*rows+j) = 0.00232;
	}
	    for(j=0;j<columns;j++)
	        *(b +j) = 1.21;
	        
					/////////////////////////////////////////////////////	
			///////////////////////////	DEVICE ALLOCATIONS 	//////////////////////////
					//////////////////////////////////////////////////

	hipMalloc((void**)&dev_A, sizeof(float)*rows*columns );
	hipMalloc((void**)&dev_b, sizeof(float)*columns);
	hipMalloc((void**)&dev_c, sizeof(float)*rows);
    hipMalloc((void**)&N_tds, sizeof(int));
    hipMalloc((void**)&N_bls, sizeof(int));
    hipMalloc((void**)&dev_cs, sizeof(int));
    hipMalloc((void**)&dev_rs, sizeof(int));

					/////////////////////////////////////////////////////
			/////////////////////	 COPY FROM HOST TO DEVICE   ///////////////////////////
					////////////////////////////////////////////////////////

	hipMemcpy(dev_A, A, sizeof(float)*rows*columns , hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, sizeof(float)*columns, hipMemcpyHostToDevice);
	hipMemcpy(dev_c, c, sizeof(float)*rows, hipMemcpyHostToDevice);
	hipMemcpy(N_tds, &N_ts, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(N_bls, &N_bs, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_cs, &r_s, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_rs, &c_s, sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

    
	
				//////////////////////////////////////////////////////////////
			//////////////////#########	CALL KERNEL non coalesced##########///////////////////////////
				/////////////////////////////////////////////////////////////

    clock_gettime(CLOCK_MONOTONIC,&start1);
	mult_mat_vectdiag<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timediag1: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    
    hipMemcpy(c, dev_c, sizeof(float)*rows, hipMemcpyDeviceToHost);    
    hipDeviceSynchronize();
    
	mult_mat_vectcol<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();    
	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timecol1: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    
	mult_mat_vectrow<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
    hipDeviceSynchronize();    
    clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timerow1: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
       
	mult_mat_vectdiag<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timediag2: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);

  	mult_mat_vectcol<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
  	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timecol2: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    
	mult_mat_vectrow<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
   	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timerow2: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    
	mult_mat_vectdiag<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end1);    
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timediag3: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    
	mult_mat_vectcol<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timecol3: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);
    
	mult_mat_vectrow<<<N_b,N_t>>>( dev_A,dev_b,dev_c, dev_rs,dev_cs,N_bls, N_tds);
	hipDeviceSynchronize();
	clock_gettime(CLOCK_MONOTONIC, &end1);
    time_taken = (double)((end1.tv_sec-start1.tv_sec)*1e9 + end1.tv_nsec - start1.tv_nsec);
	printf("Non sequential timerow3: %lf \n", time_taken);
    clock_gettime(CLOCK_MONOTONIC,&start1);

				/////////////////////////////////////////////////////////////
			/////////////////////		COPY RESULTS 	////////////////////////////////////////////
				//////////////////////////////////////////////////////////

    
	hipFree(dev_A);
	hipFree(dev_b);
	hipFree(dev_c);
    
    for(i=0;i<columns;i+=2000)
    printf("%d %lf \n",i+1,c[i]);
 
    return 0;

}
